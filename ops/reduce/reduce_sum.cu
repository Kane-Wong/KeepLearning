#include "hip/hip_runtime.h"
#include "common.cuh"

void _cpu_compute(float *input, float *out, int token_num)
{
    float *anc;
    for (int i = 0; i < token_num; i++)
    {
        anc = input + i * token_num;
        // std::cout << *anc << ' ' << i << std::endl;
        for (int j = 0; j < token_num; j++)
        {
            out[i] += anc[j];
        }
    }
}

__global__ void reduce_kernel_fun0(float *input, float *output)
{
    int tid = threadIdx.x;
    float *x = input + blockDim.x * blockIdx.x;
    for (unsigned int s = 1; s < blockDim.x; s *= 2)
    {
        if (tid % (2 * s) == 0)
        {
            // if(0 == blockIdx.x){
            //     printf("tid: %d, tid_s: %d, a: %f, b: %f \n", tid, tid+s, x[tid], x[tid+s]);
            // }
            x[tid] += x[tid + s];
        }
        __syncthreads();
    }
    if (tid == 0)
        output[blockIdx.x] = x[0];
}

template <int BLOCK_SIZE>
__global__ void reduce_kernel_fun1(float *input, float *output)
{
    __shared__ float sdata[BLOCK_SIZE];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    sdata[tid] = input[i];
    __syncthreads();
    for (unsigned int s = 1; s < blockDim.x; s *= 2)
    {
        if (tid % (2 * s) == 0)
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    if (tid == 0)
        output[blockIdx.x] = sdata[0];
}

template <int BLOCK_SIZE>
__global__ void reduce_kernel_fun2(float *input, float *output)
{
    __shared__ float sdata[BLOCK_SIZE];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    sdata[tid] = input[i];
    __syncthreads();
    for (unsigned int s = 1; s < blockDim.x; s *= 2)
    {
        if (tid * 2 * s < blockDim.x)
        {
            int index = tid * 2 * s; 
            sdata[index] += sdata[index + s];
        }
        __syncthreads();
    }
    if (tid == 0)
        output[blockIdx.x] = sdata[0];
}

template <int BLOCK_SIZE>
__global__ void reduce_kernel_fun3(float *input, float *output)
{
    __shared__ float sdata[BLOCK_SIZE];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    sdata[tid] = input[i];
    __syncthreads();
    for (unsigned int s = blockDim.x / 2; s > 0; s /= 2)
    {
        if (tid < s)
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    if (tid == 0)
        output[blockIdx.x] = sdata[0];
}

// 计算与访存并行， BLOCK_SIZE 为elements numbers的一半
template <int BLOCK_SIZE>
__global__ void reduce_kernel_fun4(float *input, float *output)
{
    __shared__ float sdata[BLOCK_SIZE];
    int tid = threadIdx.x;
    int i = blockIdx.x * 2 * blockDim.x + threadIdx.x;
    sdata[tid] = input[i] + input[i + blockDim.x];      
    __syncthreads();
    for (unsigned int s = blockDim.x / 2; s > 0; s /= 2)
    {
        if (tid < s)
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    if (tid == 0)
        output[blockIdx.x] = sdata[0];
}

template <int BLOCK_SIZE>
__global__ void reduce_kernel_fun5(float *input, float *output)
{
    volatile __shared__ float sdata[BLOCK_SIZE];
    int tid = threadIdx.x;
    int i = blockIdx.x * 2 * blockDim.x + threadIdx.x;
    sdata[tid] = input[i] + input[i + blockDim.x];      
    __syncthreads();
    for (unsigned int s = blockDim.x / 2; s > 32; s /= 2)
    {
        if (tid < s)
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    if (tid < 32)
    {
        sdata[tid] += sdata[tid + 32];
        sdata[tid] += sdata[tid + 16];
        sdata[tid] += sdata[tid + 8];
        sdata[tid] += sdata[tid + 4];
        sdata[tid] += sdata[tid + 2];
        sdata[tid] += sdata[tid + 1];
    }
    if (tid == 0)
        output[blockIdx.x] = sdata[0];
}

__global__ void reduce_kernel_fun6(float *input, float *output, const int M, const int N)
{
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    for(int i = bid; i < M; i += gridDim.x){
        float* row_pointer = input + i * N;
        float sum_val = 0;
        for(int j = tid; j < N; j += warpSize)
        {
            sum_val += row_pointer[j];
        }
        for(int offset = warpSize / 2; offset > 0; offset >>= 1)
        {
            sum_val += __shfl_xor_sync(0xffffffff, sum_val, offset);
        }
        output[i] = sum_val;
    }
}

int main(int argc, char **argv){
    int cpu_case = atoi(argv[1]);
    int gpu_case = atoi(argv[2]);
    printf("cpu running case %d, gpu running case %d\n", cpu_case, gpu_case);

    int deviceID = 0;
    hipSetDevice(deviceID);
    printDeviceInfor(deviceID);
    
    // 1. 环境准备
    const int vector_size = 1024;
    size_t data_size = vector_size * vector_size * sizeof(float);
    size_t result_size = vector_size * sizeof(float);
    float *a = (float *)malloc(data_size);
    float *r_c = (float *)malloc(result_size);
    float *r_g = (float *)malloc(result_size);

    for (int i=0; i<vector_size*vector_size; i++){
        a[i] = i%10;
    }
    for (int i=0; i<vector_size; i++){
        r_c[i] = 0;
        r_g[i] = 0;
    }
    
    float *d_a, *d_r;
    hipMalloc((void **)&d_a, data_size);
    hipMalloc((void **)&d_r, result_size);
    hipMemcpy(d_a, a, data_size, hipMemcpyHostToDevice);

    // 2. gpu 计算
    timerecord start, end;
    start.freshtime();
    switch (gpu_case)
    {
    case 0:
        reduce_kernel_fun0<<<dim3(vector_size), dim3(vector_size)>>>(d_a, d_r);
        break;
    case 1:
        reduce_kernel_fun1<vector_size><<<dim3(vector_size), dim3(vector_size)>>>(d_a, d_r);
        break;
    case 2:
        reduce_kernel_fun2<vector_size><<<dim3(vector_size), dim3(vector_size)>>>(d_a, d_r);
        break;
    case 3:
        reduce_kernel_fun3<vector_size><<<dim3(vector_size), dim3(vector_size)>>>(d_a, d_r);
        break;
    case 4:
        reduce_kernel_fun4<vector_size/2><<<dim3(vector_size), dim3(vector_size/2)>>>(d_a, d_r);
        break;
    case 5:
        reduce_kernel_fun5<vector_size/2><<<dim3(vector_size), dim3(vector_size/2)>>>(d_a, d_r);
        break;
    case 6:
        reduce_kernel_fun6<<<dim3(vector_size), dim3(32)>>>(d_a, d_r, vector_size, vector_size);
        break;
    default:
        printf("Error: Invalid gpu running case: %d\n", gpu_case);
        return EXIT_FAILURE;
    }    
    hipDeviceSynchronize(); 
    hipMemcpy(r_g, d_r, result_size, hipMemcpyDeviceToHost);
    end.freshtime();
    printTimeGap(start, end, "gpu");

    // 3. cpu 计算
    start.freshtime();
    switch (cpu_case)
    {
    case 0:
        _cpu_compute(a, r_c, vector_size);
        break;
    default:
        printf("Error: Invalid cpu running case: %d\n", cpu_case);
        return EXIT_FAILURE;
    }
    end.freshtime();
    printTimeGap(start, end, "cpu");
    
    // 4. 结果验证
    validResult<float>(r_c, r_g, vector_size, 1e-10);

    // 5. 环境释放
    free(a);
    free(r_c);
    free(r_g);
    hipHostFree(d_a);
    hipHostFree(d_r);
    return 0;
}