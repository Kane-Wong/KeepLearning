#include "hip/hip_runtime.h"
#include "common.cuh"
using namespace std;

void _cpu_compute_fun0(float *a, float *b, float *r, int matrix_size){
    for (int i=0; i<matrix_size; i++){
        for (int j=0; j<matrix_size; j++){
            for (int k=0; k<matrix_size; k++){
                r[i*matrix_size+j] += a[i*matrix_size+k] * b[k*matrix_size+j];
            }
        }
    }
}

void _cpu_compute_fun1(float *a, float *b, float *r, int matrix_size){
    for (int i=0; i<matrix_size; i++){
        for (int j=0; j<matrix_size; j++){
            float sum=0.0;
            for (int k=0; k<matrix_size; k++){
                sum += a[i*matrix_size+k] * b[k*matrix_size+j];
            }
            r[i*matrix_size+j] = sum;
        }
    }
}

// 每次计算的读/写均从全局内存取数据
__global__ void _cuda_compute_fun0(float *a, float *b, float *r, int matrix_size){
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    // printf("%d, %d, %d, %d\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y);
    if (x<matrix_size && y<matrix_size){
        for (int k=0; k<matrix_size; k++){
            r[y*matrix_size+x] += a[y*matrix_size+k] * b[k*matrix_size+x];
        }
    }
}

// 每次循环计算的sum作为局部变量存在寄存器中
__global__ void _cuda_compute_fun1(float *a, float *b, float *r, int matrix_size){
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    if (x<matrix_size && y<matrix_size){
        float sum=0.0;
        for (int k=0; k<matrix_size; k++){
            sum += a[y*matrix_size+k] * b[k*matrix_size+x];
        }
        r[y*matrix_size+x] = sum;
    }
}

// 沿K维度切成若干份，每次计算将每份数据由全局内存加载至共享内存中
template<const int BLOCK_SIZE_X, const int BLOCK_SIZE_Y>
__global__ void _cuda_compute_fun2(float *a, float *b, float *r, int matrix_size)
{
    const int blockK = 32;
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;
    __shared__ float matA[BLOCK_SIZE_Y][blockK];
    __shared__ float matB[blockK][BLOCK_SIZE_X];
    assert(BLOCK_SIZE_X == BLOCK_SIZE_Y && BLOCK_SIZE_X == blockK);
    float sumVal = 0.0f;
    for(int i=0; i<matrix_size/blockK; i++)
    {
        // 利用线程阵列将数据从全局内存放置共享内存中，需要 blockK = BLOCK_SIZE_X = BLOCK_SIZE_Y
        matA[threadIdx.y][threadIdx.x] = *(a + y * matrix_size + i * blockK + threadIdx.x);
        matB[threadIdx.y][threadIdx.x] = *(b + (i * blockK + threadIdx.y) * matrix_size + x);
        __syncthreads();
        
        for(int j=0; j<blockK; j++)
        {
            sumVal += matA[threadIdx.y][j] * matB[j][threadIdx.x];
        }
        __syncthreads();
    }
    r[y*matrix_size+x] = sumVal;
}

template<const int BLOCK_SIZE_X, const int BLOCK_SIZE_Y>
__global__ void _cuda_compute_fun3(float *a, float *b, float *r, int matrix_size)
{
    const int blockK = 8;
    const int threadM = 8;
    const int threadN = 8;
    const int x = (blockDim.x * blockIdx.x + threadIdx.x) * threadN;
    const int y = (blockDim.y * blockIdx.y + threadIdx.y) * threadM;
    __shared__ float matA[BLOCK_SIZE_Y][blockK];
    __shared__ float matB[blockK][BLOCK_SIZE_X];
    float threaA[threadM], threaB[threadN];
    float threaR[threadM][threadN]={0.0f};

    for(int i=0; i<matrix_size/blockK; i++)
    {
        float* matAblock = a + (blockDim.y * blockIdx.y) * threadM * matrix_size + i * blockK;
        float* matBblock = b + i * blockK * matrix_size + (blockDim.x * blockIdx.x) * threadN;

        for(int m=0; m<BLOCK_SIZE_Y; m+=threadM)
        {
            for(int n=0; n<blockK; n+=blockDim.x)
            {
                matA[threadIdx.y+m][threadIdx.x+n] = *(matAblock + (threadIdx.y+m) * matrix_size + threadIdx.x+n);
                matB[threadIdx.y+n][threadIdx.x+m] = *(matBblock + (threadIdx.y+n) * matrix_size + threadIdx.x + m);
            }
        }
        __syncthreads();

        for(int j=0; j<blockK; j++)
        {
            for(int k=0; k<threadM; k++)
            {
                threaA[k] = matA[threadM * threadIdx.y + k][j];
            }
            for(int k=0; k<threadN; k++)
            {
                threaB[k] = matB[j][threadN * threadIdx.x + k];
            }
            for(int m=0; m<threadM; m++)
            {
                for(int n=0; n<threadN; n++)
                {
                    threaR[m][n] += threaA[m]*threaB[n];
                }
            }
        }
        __syncthreads();
    }
    
    for(int m=0; m<threadM; m++)
    {
        for(int n=0; n<threadN; n++)
        {
            r[(y+m)*matrix_size+x+n] = threaR[m][n];
        }
    }
}

int main(int argc, char **argv)
{
    int cpu_case = atoi(argv[1]);
    int gpu_case = atoi(argv[2]);
    printf("cpu running case %d, gpu running case %d\n", cpu_case, gpu_case);
    
    int deviceID = 0;
    hipSetDevice(deviceID);
    printDeviceInfor(deviceID);
    
    // 1. 环境准备
    const int matrix_size = 1024;
    size_t size = matrix_size * matrix_size * sizeof(float);
    float *a = (float *)malloc(size);
    float *b = (float *)malloc(size);
    float *r = (float *)malloc(size);
    float *r_re = (float *)malloc(size);
    for(int i=0; i<matrix_size*matrix_size; i++){
        a[i] = rand() % matrix_size;
        b[i] = rand() % matrix_size;
        r[i] = 0;
        r_re[i] = 0;
    }

    float *d_a, *d_b, *d_r;
    hipMalloc((void **) &d_a, size);
    hipMalloc((void **) &d_b, size);
    hipMalloc((void **) &d_r, size);
    const size_t block_x=64, block_y=64;
    size_t grid_x = (matrix_size+block_x -1) / block_x;
    size_t grid_y = (matrix_size+block_y-1) / block_y;
    dim3 dimGrid(grid_y, grid_x);
    dim3 dimBlock(block_y, block_x);
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // 2. gpu 计算
    timerecord start, end;
    start.freshtime();
    switch (gpu_case)
    {
    case 0:
        _cuda_compute_fun0<<<dimGrid, dimBlock>>>(d_a, d_b, d_r, matrix_size);
        break;
    case 1:
        _cuda_compute_fun1<<<dimGrid, dimBlock>>>(d_a, d_b, d_r, matrix_size);
        break;
    case 2:
        _cuda_compute_fun2<block_x, block_y><<<dimGrid, dimBlock>>>(d_a, d_b, d_r, matrix_size);
        break;
    case 3:
        _cuda_compute_fun3<block_x, block_y><<<dimGrid, dim3(block_y/8, block_x/8)>>>(d_a, d_b, d_r, matrix_size); 
        break;
    default:
        printf("Error: Invalid gpu running case: %d\n", gpu_case);
        return EXIT_FAILURE;
    }
    hipDeviceSynchronize(); // 同步，且检查执行期间发生的错误
    hipMemcpy(r_re, d_r, size, hipMemcpyDeviceToHost);
    end.freshtime();
    printTimeGap(start, end, "gpu");
    
    // 3. cpu 计算
    start.freshtime();
    switch (cpu_case)
    {
    case 0:
        _cpu_compute_fun0(a, b, r, matrix_size);
        break;
    case 1:
        _cpu_compute_fun1(a, b, r, matrix_size);
        break;
    default:
        printf("Error: Invalid cpu running case: %d\n", cpu_case);
        return EXIT_FAILURE;
    }
    end.freshtime();
    printTimeGap(start, end, "cpu");

    // 4. 结果验证
    validResult<float>(r, r_re, matrix_size*matrix_size, 1e-10);

    // 5. 环境释放
    free(a);
    free(b);
    free(r);
    free(r_re);
    hipHostFree(d_a);
    hipHostFree(d_b);
    hipHostFree(d_r);
   
    return 0;
}