#include "common.h"

void printDeviceInfor(int deviceID)
{
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, deviceID);
    std::cout << "GPU device " << deviceID << ": " << prop.name << std::endl;
}